/*
 ============================================================================
 Proyecto GPU
 Realizado por: Anabel Díaz Labrador

 ============================================================================
 */


#include <iostream>

#include <hip/hip_runtime.h>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

void show_vector(int*, int, int);

#define N (1048576)
#define M (8)
#define HILOSPORBLOQUE (512)


// Device kernel
__global__ void incHist(const int *A, int numElements, int *histogram, int numElementsHistogram) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (i < numElements) {
    int increment = A[i] % M;
    atomicAdd(&histogram[M * blockIdx.x + increment], 1);
  }
}

__global__ void reduccion_paralela(int *histogram, int numElements, int *result) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements) {
    int middle = numElements / 2;
    while (middle >= M) {  // Hacemos reducción hasta que queden por juntar 8. Ultima iteración middle = 4
      if (i < middle) {
        histogram[i] = histogram[i] + histogram[i + middle];
      }
      __syncthreads();
      middle = middle / 2;
    } 
  } 

  if (i >= 0 && i < M) {
    result[i] = histogram[i];
  }
}




// Host main
int main(void) {
  // Vector length to be used, and compute its size
  const int numElementsA = N;
  size_t sizeA = numElementsA * sizeof(int);

  // Allocate the host input vector A
  int *h_A = (int*)malloc(sizeA);

  // Verify that allocations succeeded
  if (h_A == NULL) {
      fprintf(stderr, "Failed to allocate host vectors!\n");
      exit(EXIT_FAILURE);
  }

  // Initialize the host input vector with [0, 1000000] random range
  time_t t;
  srand((unsigned) time(&t));
  for (int i = 0; i < numElementsA; i++) {
    h_A[i] = rand() % N;
  }
  printf("Vector element number: %d\n", numElementsA);
  //show_vector(h_A, 0, 10); Comprobamos que añade números aleatorios

  // Allocate the device input vector A
  int *d_A = NULL;
  CUDA_CHECK_RETURN(hipMalloc((void**)&d_A, sizeA));

  // Copy the host input vector A in host memory to the device input vector in
  // device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  CUDA_CHECK_RETURN(hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice));

  // Calculamos el número de bloques necesario
  int threadsPerBlock = HILOSPORBLOQUE;
  int blocksPerGrid = (numElementsA + threadsPerBlock - 1) / threadsPerBlock;

  // Vector length to be used, and compute its size
  int numElementsHistogram = blocksPerGrid * M;
  size_t sizeHistogram = numElementsHistogram * sizeof(int);

  // Allocate the host input vector histogram
  int *h_histograms = (int*)malloc(sizeHistogram);
  if (h_histograms == NULL) {
      fprintf(stderr, "Failed to allocate host vectors!\n");
      exit(EXIT_FAILURE);
  }

  // Initialize the host input vector
  for (int i = 0; i < numElementsHistogram; i++) {
    h_histograms[i] = 0;
  }

  // Allocate the device input vector histogram
  int *d_histograms = NULL;
  CUDA_CHECK_RETURN(hipMalloc((void**)&d_histograms, sizeHistogram));

  // Copy the host input vector histograms in host memory to the device input vector in
  // device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  CUDA_CHECK_RETURN(hipMemcpy(d_histograms, h_histograms, sizeHistogram, hipMemcpyHostToDevice));

  // Launch the incHist CUDA Kernel
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  incHist<<<blocksPerGrid, threadsPerBlock>>>(d_A, numElementsA, d_histograms, numElementsHistogram);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  CUDA_CHECK_RETURN(hipGetLastError());

  float elapsedTime1;
  hipEventElapsedTime(&elapsedTime1, start, stop);
  
  // Recuperamos histogramas
  printf("Copy local histograms from the CUDA device to the host memory\n");
  CUDA_CHECK_RETURN(hipMemcpy(h_histograms, d_histograms, sizeHistogram, hipMemcpyDeviceToHost));

  // Checkeamos vector
  show_vector(h_histograms, 0, M);
  int acc = 0;
  for (int i = 0; i < numElementsHistogram; i++) {
    acc += h_histograms[i];
  }
  printf("Histogram total increments: %d\nHistogram size: %d\n", acc, numElementsHistogram);

  // Allocate the host input vector histograma
  int *h_histogram = (int*)malloc((size_t)(M * sizeof(int)));
  if (h_histogram == NULL) {
      fprintf(stderr, "Failed to allocate host vectors!\n");
      exit(EXIT_FAILURE);
  }

  // Allocate the device input vector histogram
  int *d_histogram = NULL;
  CUDA_CHECK_RETURN(hipMalloc((void**)&d_histogram, (size_t)(M * sizeof(int))));

  // Initialize the host input vector
  for (int i = 0; i < M; i++) {
    h_histogram[i] = 0;
  }

  // Copy the host input vector histogram in host memory to the device input vector in
  // device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  CUDA_CHECK_RETURN(hipMemcpy(d_histogram, h_histogram,  (size_t)(M * sizeof(int)), hipMemcpyHostToDevice));

  // Launch the reduccion_paralela CUDA Kernel
  blocksPerGrid = (numElementsHistogram + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  hipEventRecord(start, 0);
  reduccion_paralela<<<blocksPerGrid, threadsPerBlock>>>(d_histograms, numElementsHistogram, d_histogram);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  CUDA_CHECK_RETURN(hipGetLastError());

  float elapsedTime2;
  hipEventElapsedTime(&elapsedTime2, start, stop);

  // Recuperamos histograma resultado
  printf("Copy histogram result from the CUDA device to the host memory\n");
  CUDA_CHECK_RETURN(hipMemcpy(h_histogram, d_histogram, (size_t)(M * sizeof(int)), hipMemcpyDeviceToHost));

  // Checkeamos vector
  show_vector(h_histogram, 0, M);
  acc = 0;
  for (int i = 0; i < M; i++) {
    acc += h_histogram[i];
  }
  printf("Histogram total data: %d\n", acc);

  // Free device global memory
  CUDA_CHECK_RETURN(hipFree(d_A));
  CUDA_CHECK_RETURN(hipFree(d_histograms));
  CUDA_CHECK_RETURN(hipFree(d_histogram));

  // Free host memory
  free(h_A);
  free(h_histograms);
  free(h_histogram);

  printf("Tiempo construyendo histogramas locales: %f milisegundos\n", elapsedTime1);
  printf("Tiempo juntando histogramas en uno final: %f milisegundos\n", elapsedTime2);
  printf("Tiempo total: %f milisegundos\n", elapsedTime1 + elapsedTime2);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  printf("Done\n");
  return EXIT_SUCCESS;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err) {

	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (EXIT_FAILURE);
}


void show_vector(int* vector, int min, int max) {
  printf("[%d", vector[min]);
  for (unsigned i = min + 1; i < max; i++) 
    printf(", %d", vector[i]);   
  printf("]\n");
}